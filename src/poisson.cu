#include "cuda_globals.hpp"

#include <iostream>
#include <fstream>

template<typename T>
float run_cuda( const Parameters& _parameters )
{
  Data<T> data;
  hiprandGenerator_t gen;
  hipEvent_t custart, cuend;
  float ms=0.f;

  data.poisson_numbers_h = new T[_parameters.n];

  CHECK_CUDA( hipEventCreate(&custart) );
  CHECK_CUDA( hipEventCreate(&cuend) );

  CHECK_CUDA(hiprandCreateGenerator(&gen,
                                    HIPRAND_RNG_PSEUDO_DEFAULT));
  /* Set seed */
  CHECK_CUDA(hiprandSetPseudoRandomGeneratorSeed(
                gen, 1234ULL));

  /* Allocate n unsigned ints on device */
  CHECK_CUDA(hipMalloc(&data.poisson_numbers_d,
                       _parameters.n * sizeof(T)));

  CHECK_CUDA(hipEventRecord(custart));

  /* Generate n unsigned ints on device */
  CHECK_CUDA(hiprandGeneratePoisson(gen,
                                    data.poisson_numbers_d,
                                    _parameters.n,
                                    _parameters.lambda));
  CHECK_CUDA(hipEventRecord(cuend));

  CHECK_CUDA( hipEventSynchronize(cuend) );
  CHECK_CUDA( hipEventElapsedTime(&ms, custart, cuend) );

  CHECK_CUDA( hipMemcpy(data.poisson_numbers_h, data.poisson_numbers_d, _parameters.n*sizeof(T), hipMemcpyDeviceToHost) );

  CHECK_CUDA( hipEventDestroy(custart) );
  CHECK_CUDA( hipEventDestroy(cuend) );
  CHECK_CUDA( hipFree(data.poisson_numbers_d) );


  std::ofstream fs;

  fs.open("dump.csv", std::ofstream::out);
  for( auto i=0; i<_parameters.n; ++i ) {
    fs << data.poisson_numbers_h[i] << std::endl;
  }
  fs.close();
  std::cout <<_parameters.n<< " Poisson numbers dumped to dump.csv." << std::endl;
  delete[] data.poisson_numbers_h;

  return ms;
}

int main(int argc, char** argv)
{
  Parameters parameters;
  if(argc>=2)
    parameters.n = atoi(argv[1]);
  if(argc==3)
    parameters.lambda = atof(argv[2]);
  float ms = run_cuda<unsigned>(parameters);
  std::cout << std::endl << parameters.n << " Poisson numbers with lambda = " << parameters.lambda << std::endl;
  std::cout << " ... generated on device in: " << ms << " ms" << std::endl;
  return 0;
}